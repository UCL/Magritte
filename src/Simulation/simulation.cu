#include "Simulation/simulation.hpp"
#include "Raypair/raypair.cuh"
#include "Raypair/rayblock.cuh"


int Simulation :: handleCudaError (hipError_t error)
{
    if (error != hipSuccess)
    {
        logger.write ("CUDA ERROR : " + string (hipGetErrorString (error)));
    }

    return (0);
}


int Simulation :: gpu_get_device_properties (void)
{
    int nDevices;
    HANDLE_ERROR (hipGetDeviceCount (&nDevices));

    logger.write_line (                                      );
    logger.write      (" Properties of the available GPU's :");
    logger.write_line (                                      );

    for (long i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t prop;
        HANDLE_ERROR (hipGetDeviceProperties (&prop, i));

        const string cr  = std::to_string(prop.memoryClockRate);
        const string bw  = std::to_string(prop.memoryBusWidth);
        const string cc  = std::to_string(prop.major)+"."+std::to_string(prop.minor);
        const double pmb = 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6;

        logger.write_line (                                                     );
        logger.write      ("Device Number                : ",  i                );
        logger.write      ("Device name                  : " + string(prop.name));
        logger.write      ("Compute compatibility        : " + cc               );
        logger.write      ("Memory Clock Rate (KHz)      : " + cr               );
        logger.write      ("Memory Bus Width (bits)      : " + bw               );
        logger.write      ("Peak Memory Bandwidth (GB/s) : ",  pmb              );
        logger.write_line (                                                     );
    }

    return (0);
}


//int Simulation :: gpu_compute_radiation_field (void)
//{
//  // Initialisations
//  for (LineProducingSpecies &lspec : lines.lineProducingSpecies)
//  {
//    lspec.lambda.clear ();
//  }
//
//  radiation.initialize_J ();
//
//  /// Set maximum number of points along a ray, if not set yet
//  if (geometry.max_npoints_on_rays == -1)
//  {
//    get_max_npoints_on_rays <CoMoving> ();
//  }
//
//  /// Create a gpuRayPair object
//  gpuRayPair *raypair = new gpuRayPair (geometry.max_npoints_on_rays,
//                                        parameters.ncells(),
//                                        parameters.nfreqs(),
//                                        parameters.nlines()          );
//
//  /// Set model data
//  raypair->copy_model_data (Simulation(*this));
//
//
//  for (long r = 0; r < parameters.nrays()/2; r++)
//  {
//    const long R = r - MPI_start (parameters.nrays()/2);
//
//    logger.write ("ray = ", r);
//
//    for (long o = 0; o < parameters.ncells(); o++)
//    {
//      const long           ar = geometry.rays.antipod[r];
//      const double weight_ang = geometry.rays.weights[r];
//      const double dshift_max = get_dshift_max (o);
//
//
//      // Trace ray pair
//      const RayData raydata_r  = geometry.trace_ray <CoMoving> (o, r,  dshift_max);
//      const RayData raydata_ar = geometry.trace_ray <CoMoving> (o, ar, dshift_max);
//
//      if (raydata_r.size() + raydata_ar.size() > 0)
//      {
//        /// Setup such that the first ray is the longest (for performance)
//        raypair->setup (*this, raydata_ar, raydata_r, R, o);
//        /// Solve radiative transfer along ray pair
//        raypair->solve ();
//        /// Extract model data
//        raypair->extract_radiation_field (*this, R, r, o);
//      }
//      else
//      {
//        /// Extract radiation field from boundary consitions
//        get_radiation_field_from_boundary (R, r, o);
//      }
//    }
//  }
//
//  /// Delete raypair
//  delete raypair;
//
//  return (0);
//}


int Simulation :: cpu_compute_radiation_field (const double inverse_dtau_max)
{
    // Initialisations
    for (LineProducingSpecies &lspec : lines.lineProducingSpecies)
    {
        lspec.lambda.clear ();
    }

    radiation.initialize_J ();

    /// Set maximum number of points along a ray, if not set yet
    if (geometry.max_npoints_on_rays == -1)
    {
        get_max_npoints_on_rays <CoMoving> ();
    }

    const long nraypairs = 1;

    /// Create a gpuRayPair object
    RayBlock_v *rayblock = new RayBlock_v (parameters.ncells(),
                                           parameters.nfreqs(),
                                           parameters.nlines(),
                                           nraypairs,
                                           geometry.max_npoints_on_rays);

    /// Set inverse maximum optical depth increment
    rayblock->inverse_dtau_max = inverse_dtau_max;

    /// Set model data
    rayblock->copy_model_data (*this);


    for (size_t rr = 0; rr < parameters.nrays()/2; rr++)
    {
        const size_t RR = rr - MPI_start (parameters.nrays()/2);
        const size_t ar = geometry.rays.antipod[rr];

        logger.write ("ray = ", rr);

        for (size_t o = 0; o < parameters.ncells(); o++)
        {
            const double dshift_max = get_dshift_max (o);

            // Trace ray pair
            const RayData ray_ar = geometry.trace_ray <CoMoving> (o, ar, dshift_max);
            const RayData ray_rr = geometry.trace_ray <CoMoving> (o, rr, dshift_max);

            const size_t depth = ray_ar.size() + ray_rr.size() + 1;

            if (depth > 1)
            {
                const ProtoRayBlock_v prb (ray_ar, ray_rr, o);

                rayblock->setup (*this, RR, rr, ProtoRayBlock_v (ray_ar, ray_rr, o));
                rayblock->solve ();
                rayblock->store (*this);
            }
            else
            {
                /// Extract radiation field from boundary
                get_radiation_field_from_boundary (RR, rr, o);
            }
        }
    }

    /// Delete ray block
    delete rayblock;

    return (0);
}




//int Simulation :: gpu_compute_radiation_field_2 (
//        const size_t nraypairs,
//        const size_t gpuBlockSize,
//        const size_t gpuNumBlocks,
//        const double inverse_dtau_max             )
//{
//    // Set timers
//    Timer timer("GPU compute radiation field");
//    timer.start();
//
//    // Initialisations
//    for (LineProducingSpecies &lspec : lines.lineProducingSpecies)
//    {
//        lspec.lambda.clear ();
//    }
//
//    radiation.initialize_J ();
//
//    /// Set maximum number of points along a ray, if not set yet
//    if (geometry.max_npoints_on_rays == -1)
//    {
//        get_max_npoints_on_rays <CoMoving> ();
//    }
//
//    // Get number of threads
//    const size_t nthreads = get_nthreads();
//
//    vector<RayBlock*> rayblocks (nthreads);
//
//    for (auto &rayblock : rayblocks)
//    {
//        // Create a RayBlock object
//        rayblock = new RayBlock (parameters.ncells(),
//                                 parameters.nfreqs(),
//                                 parameters.nlines(),
//                                 nraypairs,
//                                 geometry.max_npoints_on_rays);
//
//        /// Set GPU block size
//        rayblock->gpuBlockSize     = gpuBlockSize;
//        rayblock->gpuNumBlocks     = gpuNumBlocks;
//        rayblock->inverse_dtau_max = inverse_dtau_max;
//
//        /// Set model data
//        rayblock->copy_model_data (*this);
//    }
//
//
//    for (size_t rr = 0; rr < parameters.nrays()/2; rr++)
//    {
//        const size_t RR = rr - MPI_start (parameters.nrays()/2);
//        const size_t ar = geometry.rays.antipod[rr];
//
//        RayQueue rayqueue (nraypairs);
//
////        cout << "complete = ";
////        if (rayqueue.complete()) {cout << "True"  << endl;}
////        else                     {cout << "False" << endl;}
//
//
//        logger.write ("ray = ", rr);
//
////#       pragma omp parallel default (shared)
//        {
////            const size_t t = omp_get_thread_num();
//            auto &rayblock = rayblocks[omp_get_thread_num()];
//
//
//            for (size_t o = omp_get_thread_num(); o < parameters.ncells(); o += omp_get_num_threads())
//            {
//                const double dshift_max = get_dshift_max (o);
//
//                // Trace ray pair
//                const RayData ray_ar = geometry.trace_ray <CoMoving> (o, ar, dshift_max);
//                const RayData ray_rr = geometry.trace_ray <CoMoving> (o, rr, dshift_max);
//
//                const size_t depth = ray_ar.size() + ray_rr.size() + 1;
//
//                if (depth > 1)
//                {
//#                   pragma omp critical (add_to_queue)
//                    {
//                        /// Add ray pair to queue
//                        rayqueue.add (ray_ar, ray_rr, o, depth);
//                    }
//
//#                   pragma omp critical (offload_to_gpu)
//                    {
//                        if (rayqueue.some_are_completed())
//                        {
//                            rayblock->solve_cpu (rayqueue.get_complete_block(), RR, rr, *this);
//                        }
//                    }
//                }
//                else
//                {
//                    /// Extract radiation field from boundary
//                    get_radiation_field_from_boundary (RR, rr, o);
//                }
//            }
//        }
//
//        /// Compute the unfinished rays in the queue
////        for (long s = omp_get_thread_num(); s < rayqueue.queue.size(); s += omp_get_num_threads())
//        for (const ProtoRayBlock &prb : rayqueue.queue)
//        {
////            const ProtoRayBlock &prb = rayqueue.queue[s];
//
//            rayblocks[0]->nraypairs = prb.nraypairs();
//            rayblocks[0]->width     = prb.nraypairs() * parameters.nfreqs();
//
//            rayblocks[0]->solve_cpu (prb, RR, rr, *this);
//        }
//    }
//
//
//    /// Delete ray blocks
//    for (auto &rayblock : rayblocks) {delete rayblock;}
//
//    // Stop timer and print results
//    timer.stop();
//    timer.print();
//
//    return (0);
//}


